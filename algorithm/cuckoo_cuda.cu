#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2016 John Tromp

// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html

#include <stdint.h>
#include <string.h>
#include "cuckoo.h"
#define printf(x...) 


#define bswap_16(value)  \
  ((((value) & 0xff) << 8) | ((value) >> 8))

#define bswap_32(value) \
  (((uint32_t)bswap_16((uint16_t)((value) & 0xffff)) << 16) | \
  (uint32_t)bswap_16((uint16_t)((value) >> 16)))

#define bswap_64(value) \
  (((uint64_t)bswap_32((uint32_t)((value) & 0xffffffff)) \
      << 32) | \
  (uint64_t)bswap_32((uint32_t)((value) >> 32)))


static inline void flip80(void *dest_p, const void *src_p)
{
  uint32_t *dest = (uint32_t *)dest_p;
  const uint32_t *src = (uint32_t *)src_p;
  int i;

  for (i = 0; i < 20; i++)
    dest[i] = bswap_32(src[i]);
}

const char *errstr[] = { "OK", "wrong header length", "nonce too big", "nonces not ascending", "endpoints don't match up", "branch in cycle", "cycle dead ends", "cycle too short"};
// d(evice s)ipnode
#if (__CUDA_ARCH__  >= 320) // redefine ROTL to use funnel shifter, 3% speed gain

static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) { return make_uint2(a.x ^ b.x, a.y ^ b.y); }
static __device__ __forceinline__ void operator^= (uint2 &a, uint2 b) { a.x ^= b.x, a.y ^= b.y; }
static __device__ __forceinline__ void operator+= (uint2 &a, uint2 b) {
  asm("{\n\tadd.cc.u32 %0,%2,%4;\n\taddc.u32 %1,%3,%5;\n\t}\n\t"
    : "=r"(a.x), "=r"(a.y) : "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
}
#undef ROTL
__inline__ __device__ uint2 ROTL(const uint2 a, const int offset) {
  uint2 result;
  if (offset >= 32) {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  } else {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
  return result;
}
__device__ __forceinline__ uint2 vectorize(const uint64_t x) {
  uint2 result;
  asm("mov.b64 {%0,%1},%2; \n\t" : "=r"(result.x), "=r"(result.y) : "l"(x));
  return result;
}
__device__ __forceinline__ uint64_t devectorize(uint2 x) {
  uint64_t result;
  asm("mov.b64 %0,{%1,%2}; \n\t" : "=l"(result) : "r"(x.x), "r"(x.y));
  return result;
}
__device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  uint2 nonce = vectorize(2*nce + uorv);
  uint2 v0 = vectorize(keys.k0),
        v1 = vectorize(keys.k1),
        v2 = vectorize(keys.k2),
        v3 = vectorize(keys.k3) ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= vectorize(0xff);
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return devectorize(v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#else

__device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  u64 nonce = 2*nce + uorv;
  u64 v0 = keys.k0, v1 = keys.k1, v2 = keys.k2, v3 = keys.k3 ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= 0xff;
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}
 
#endif

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making twice_set the
// same size as shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(cuckoo_hash) == sizeof(twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (NNODES >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// NNODES >> IDXSHIFT == NEDGES >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif

#define NODEBITS (EDGEBITS + 1)
#define NODEMASK (NNODES-1)

// grow with cube root of size, hardly affected by trimming
#define MAXPATHLEN (8 << (NODEBITS/3))

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset {
public:
  u32 *bits;
  __device__ void reset(edge_t n) {
    bits[n/32] |= 1 << (n%32);
  }
  __device__ bool test(node_t n) const {
    return !((bits[n/32] >> (n%32)) & 1);
  }
  __device__ u32 block(node_t n) const {
    return ~bits[n/32];
  }
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (NEDGES >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class twice_set {
public:
  u32 *bits;
  __device__ void reset() {
    memset(bits, 0, TWICE_WORDS * sizeof(u32));
  }
  __device__ void set(node_t u) {
    node_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit) atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = (u64 *)calloc(CUCKOO_SIZE, sizeof(u64));
    assert(cuckoo != 0);
  }
  ~cuckoo_hash() {
    free(cuckoo);
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 old = 0;
      if (cuckoo[ui].compare_exchange_strong(old, niew, std::memory_order_relaxed))
        return;
      if ((old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui].store(niew, std::memory_order_relaxed);
#else
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
#endif
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 cu = cuckoo[ui].load(std::memory_order_relaxed);
#else
      u64 cu = cuckoo[ui];
#endif
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & NODEMASK);
      }
    }
  }
};

class cuckoo_ctx {
public:
  siphash_keys sip_keys;
  shrinkingset alive;
  twice_set nonleaf;
  int nthreads;

  cuckoo_ctx(const u32 n_threads) {
    nthreads = n_threads;
  }
  void setheadernonce(char* headernonce, const u32 nonce) {
    ((u32 *)headernonce)[HEADERLEN/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, HEADERLEN, &sip_keys);
  }
};

__global__ void count_node_deg(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys; // local copy sip context; 2.5% speed gain
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (edge_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (edge_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      node_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        nonleaf.set(u >> PART_BITS);
      }
    }
  }
}

__global__ void kill_leaf_edges(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (edge_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (edge_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      node_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        if (!nonleaf.test(u >> PART_BITS)) {
          alive.reset(nonce);
        }
      }
    }
  }
}

u32 path(cuckoo_hash &cuckoo, node_t u, node_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo[u]) {
    if (nu >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        printf("maximum path length exceeded\n");
      else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      return UINT_MAX;
    }
    us[nu++] = u;
  }
  return nu-1;
}

typedef std::pair<node_t,node_t> edge;

extern "C" int cuckoo_scanhash(const char *aHeader, uint32_t nonce, uint32_t *proof) {
  int nthreads = 16384;
  int trims   = 32;
  int tpb = 0;
  int range = 1;
  u64 *bits = NULL;
  int ret = -1;
  unsigned char header[80];

  if (!tpb) // if not set, then default threads per block to roughly square root of threads
    for (tpb = 1; tpb*tpb < nthreads; tpb *= 2) ;

  //if (range > 1)
    //printf("-%d", nonce+range-1);
  //printf(") with 50%% edges, %d trims, %d threads %d per block\n", trims, nthreads, tpb);

  flip80(header, aHeader);

  cuckoo_ctx ctx(nthreads);

  char headernonce[HEADERLEN];
  u32 hdrlen = HEADERLEN-4;
  memcpy(headernonce, header, hdrlen);
  memset(headernonce+hdrlen, 0, sizeof(headernonce)-hdrlen);

  u64 edgeBytes = NEDGES/8, nodeBytes = TWICE_WORDS*sizeof(u32);
  checkCudaErrors(hipMalloc((void**)&ctx.alive.bits, edgeBytes));
  checkCudaErrors(hipMalloc((void**)&ctx.nonleaf.bits, nodeBytes));

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  //printf("Using %d%cB edge and %d%cB node memory.\n",
     //(int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);

  cuckoo_ctx *device_ctx;
  checkCudaErrors(hipMalloc((void**)&device_ctx, sizeof(cuckoo_ctx)));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  for (int r = 0; r < range; r++) {
    hipEventRecord(start, NULL);
    checkCudaErrors(hipMemset(ctx.alive.bits, 0, edgeBytes));
    ctx.setheadernonce(headernonce, nonce + r);
    /*
    printf("Cuckoo miner working on header: ");
    for (int i=0; i<HEADERLEN; i++) {
      printf("%x", (unsigned char)*(headernonce + i));
    }
    printf("\n");
    */
    hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);
    for (u32 round=0; round < trims; round++) {
      for (u32 uorv = 0; uorv < 2; uorv++) {
        for (u32 part = 0; part <= PART_MASK; part++) {
          checkCudaErrors(hipMemset(ctx.nonleaf.bits, 0, nodeBytes));
          count_node_deg<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
          kill_leaf_edges<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
        }
      }
    }
  
    bits = (u64 *)calloc(NEDGES/64, sizeof(u64));
    assert(bits != NULL);
    hipMemcpy(bits, ctx.alive.bits, (NEDGES/64) * sizeof(u64), hipMemcpyDeviceToHost);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float duration;
    hipEventElapsedTime(&duration, start, stop);
    u32 cnt = 0;
    for (int i = 0; i < NEDGES/64; i++)
      cnt += __builtin_popcountll(~bits[i]);
    u32 load = (u32)(100L * cnt / CUCKOO_SIZE);
    //printf("nonce %d: %d trims completed in %.3f seconds final load %d%%\n",
      //      nonce+r, trims, duration / 1000.0f, load);
  
    if (load >= 90) {
      printf("overloaded! exiting...\n");
      goto out;
    }
  
    cuckoo_hash cuckoo = cuckoo_hash();
    node_t us[MAXPATHLEN], vs[MAXPATHLEN];
    for (edge_t block = 0; block < NEDGES; block += 64) {
      u64 alive64 = ~bits[block/64];
      for (edge_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
        u32 ffs = __builtin_ffsll(alive64);
        nonce += ffs; alive64 >>= ffs;
        node_t u0=sipnode(&ctx.sip_keys, nonce, 0), v0=sipnode(&ctx.sip_keys, nonce, 1);
        if (u0) {
          u32 nu = path(cuckoo, u0, us), nv = path(cuckoo, v0, vs);
	  if (nu == UINT_MAX || nv == UINT_MAX) break;
          if (us[nu] == vs[nv]) {
            u32 min = nu < nv ? nu : nv;
            for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
            u32 len = nu + nv + 1;
            if (len == PROOFSIZE) {
	      printf("%4d-cycle found at %d:%d%%\n", len, 0, (u32)(nonce*100L/NEDGES));
	      ret = 0;
              printf("Solution");
              std::set<edge> cycle;
              u32 n = 0;
              cycle.insert(edge(*us, *vs));
              while (nu--)
                cycle.insert(edge(us[(nu+1)&~1], us[nu|1])); // u's in even position; v's in odd
              while (nv--)
                cycle.insert(edge(vs[nv|1], vs[(nv+1)&~1])); // u's in odd position; v's in even
              for (edge_t blk = 0; blk < NEDGES; blk += 64) {
                u64 alv64 = ~bits[blk/64];
                for (edge_t nce = blk-1; alv64; ) { // -1 compensates for 1-based ffs
                  u32 ffs = __builtin_ffsll(alv64);
                  nce += ffs; alv64 >>= ffs;
                  edge e(sipnode(&ctx.sip_keys, nce, 0), sipnode(&ctx.sip_keys, nce, 1));
                  if (cycle.find(e) != cycle.end()) {
                    printf(" %jx", (uintmax_t)nce);
		    proof[n] = nce;
                    if (PROOFSIZE > 2)
                      cycle.erase(e);
                    n++;
                  }
                  if (ffs & 64) break; // can't shift by 64
                }
              }
              assert(n==PROOFSIZE);
              printf("\n");
            }
          } else if (nu < nv) {
            while (nu--)
              cuckoo.set(us[nu+1], us[nu]);
            cuckoo.set(u0, v0);
          } else {
            while (nv--)
              cuckoo.set(vs[nv+1], vs[nv]);
            cuckoo.set(v0, u0);
          }
        }
        if (ffs & 64) break; // can't shift by 64
      }
    }
  }
out:
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipFree(device_ctx));
  checkCudaErrors(hipFree(ctx.nonleaf.bits));
  checkCudaErrors(hipFree(ctx.alive.bits));
  free(bits);
  return ret;
}
